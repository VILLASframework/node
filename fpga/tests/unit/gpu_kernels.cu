#include "hip/hip_runtime.h"
/** GPU CUDA kernel
 *
 * Author: Daniel Krebs <github@daniel-krebs.net>
 * SPDX-FileCopyrightText: 2017 Daniel Krebs <github@daniel-krebs.net>
 * SPDX-License-Identifier: Apache-2.0
 *********************************************************************************/

#include <cstdint>
#include <cstdio>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <villas/gpu.hpp>
#include <villas/fpga/ips/rtds2gpu.hpp>

__global__ void
gpu_rtds_rtt_kernel(volatile uint32_t* dataIn, volatile reg_doorbell_t* doorbellIn,
                    volatile uint32_t* dataOut, volatile villas::fpga::ip::ControlRegister* controlRegister,
                    int* run)
{
  printf("[gpu] gpu kernel go\n");

  printf("dataIn:      %p\n", dataIn);
  printf("doorbellIn:  %p\n", doorbellIn);
  printf("dataOut:     %p\n", dataOut);
  printf("control:     %p\n", controlRegister);
  printf("run:         %p\n", run);

//  *run = reinterpret_cast<bool*>(malloc(sizeof(bool)));
//  **run = true;

  uint32_t last_seq;
  while (*run) {
    // wait for data
//    printf("[gpu] wait for data, last_seq=%u\n", last_seq);
    while (not (doorbellIn->is_valid and (last_seq != doorbellIn->seq_nr)) and *run);
//      printf("doorbell: 0x%08x\n", doorbellIn->value);

    last_seq = doorbellIn->seq_nr;

//    printf("[gpu] copy data\n");
    for (size_t i = 0; i < doorbellIn->count; i++) {
      dataOut[i] = dataIn[i];
    }

    // reset doorbell
//    printf("[gpu] reset doorbell\n");
//    doorbellIn->value = 0;

//    printf("[gpu] signal go for gpu2rtds\n");
    controlRegister->ap_start = 1;
  }

  printf("kernel done\n");
}

static int* run = nullptr;

void gpu_rtds_rtt_start(volatile uint32_t* dataIn, volatile reg_doorbell_t* doorbellIn,
                        volatile uint32_t* dataOut, volatile villas::fpga::ip::ControlRegister* controlRegister)
{
  printf("run:         %p\n", run);
  if (run == nullptr) {
    run = (int*)malloc(sizeof(uint32_t));
    hipHostRegister(run, sizeof(uint32_t), 0);
  }
  printf("run:         %p\n", run);

  *run = 1;
  gpu_rtds_rtt_kernel<<<1, 1>>>(dataIn, doorbellIn, dataOut, controlRegister, run);
  printf("[cpu] kernel launched\n");
}

void gpu_rtds_rtt_stop()
{
  *run = 0;
  hipDeviceSynchronize();
}

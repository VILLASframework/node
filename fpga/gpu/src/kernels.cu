#include "hip/hip_runtime.h"
/** GPU Kernels.
 *
 * Author: Daniel Krebs <github@daniel-krebs.net>
 * SPDX-FileCopyrightText: 2017 Institute for Automation of Complex Power Systems, RWTH Aachen University
 * SPDX-License-Identifier: Apache-2.0
 *********************************************************************************/

#include <stdio.h>

#include <villas/gpu.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "kernels.hpp"

using namespace villas::gpu;


__global__ void
kernel_mailbox(volatile uint32_t *mailbox, volatile uint32_t* counter)
{
  printf("[gpu] hello!\n");
  printf("[gpu] mailbox: %p\n", mailbox);

  printf("[kernel] started\n");

  while (1) {
    if (*mailbox == 1) {
      *mailbox = 0;
      printf("[gpu] counter = %d\n", *counter);
      break;
    }
  }

  printf("[gpu] quit\n");
}

__global__ void
kernel_memcpy(volatile uint8_t* dst, volatile uint8_t* src, size_t length)
{
  while (length > 0) {
    *dst++ = *src++;
    length--;
  }
}
